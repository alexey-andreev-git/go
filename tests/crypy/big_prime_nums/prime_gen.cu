#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define NUM_ROUNDS 32
#define NUM_SEGMENTS 32 // For a 2048-bit number (32 * 64 bits)

extern "C" {

// CUDA kernel to initialize random states
__global__ void initCurand(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// Device function to add two large numbers modulo n
__device__ void addMod(uint64_t *a, uint64_t *b, uint64_t *mod, uint64_t *result) {
    uint64_t carry = 0;
    for (int i = 0; i < NUM_SEGMENTS; i++) {
        uint64_t sum = a[i] + b[i] + carry;
        result[i] = sum % mod[i];
        carry = sum / mod[i];
    }
}

// Device function to multiply two large numbers modulo n
__device__ void mulMod(uint64_t *a, uint64_t *b, uint64_t *mod, uint64_t *result) {
    uint64_t temp[NUM_SEGMENTS] = {0};

    for (int i = 0; i < NUM_SEGMENTS; i++) {
        __uint128_t carry = 0;
        for (int j = 0; j < NUM_SEGMENTS - i; j++) {
            // Perform multiplication and add carry, then apply modular reduction
            __uint128_t prod = (__uint128_t)a[i] * b[j] + temp[i + j] + carry;

            // Apply modular reduction on the product within each segment
            temp[i + j] = (uint64_t)(prod % mod[i + j]);
            carry = prod / mod[i + j];
        }
    }

    // Final reduction of temp array to fit within mod
    for (int k = 0; k < NUM_SEGMENTS; k++) {
        result[k] = temp[k] % mod[k];
    }
}

// Device function to perform modular exponentiation on large numbers
__device__ void modExp(uint64_t *base, uint64_t *exp, uint64_t *mod, uint64_t *result) {
    uint64_t temp[NUM_SEGMENTS] = {1}; // Initialize result as 1
    uint64_t baseTemp[NUM_SEGMENTS];
    for (int i = 0; i < NUM_SEGMENTS; i++) baseTemp[i] = base[i];

    for (int i = NUM_SEGMENTS * 64 - 1; i >= 0; i--) {
        mulMod(temp, temp, mod, temp); // result = (result * result) % mod

        // Debug: Print temp after squaring
        // printf("modExp: temp after squaring: %llu\n", temp[0]);

        if ((exp[i / 64] & (1ULL << (i % 64))) != 0) {
            mulMod(temp, baseTemp, mod, temp); // result = (result * base) % mod

            // Debug: Print temp after multiplication
            // printf("modExp: temp after multiplication: %llu\n", temp[0]);
        }
    }

    for (int k = 0; k < NUM_SEGMENTS; k++) {
        result[k] = temp[k] % mod[k];
    }
}

// CUDA kernel for Miller-Rabin primality test on segmented large numbers
__global__ void millerRabinTestKernel(uint64_t *numbers, int *results, hiprandState *state, int numCandidates) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numCandidates) return;

    hiprandState localState = state[idx];
    results[idx] = 1; // Assume prime initially

    uint64_t *n = &numbers[idx * NUM_SEGMENTS];
    uint64_t d[NUM_SEGMENTS], x[NUM_SEGMENTS], a[NUM_SEGMENTS];
    int r = 0;

    // Initialize d as n - 1
    for (int i = 0; i < NUM_SEGMENTS; i++) d[i] = n[i];
    d[0] -= 1;  // Subtract 1 (n - 1)

    // Factor d as d * 2^r
    while ((d[0] & 1) == 0) {
        for (int i = 0; i < NUM_SEGMENTS; i++) d[i] >>= 1;
        r++;
    }

    // Miller-Rabin rounds
    for (int round = 0; round < NUM_ROUNDS; round++) {
        // Generate a random base a in the range [2, n-2]
        for (int i = 0; i < NUM_SEGMENTS; i++) a[i] = hiprand(&localState) % n[i];
        a[0] = 2;

        modExp(a, d, n, x); // Compute x = a^d % n

        // Debug: Print x after modExp
        // printf("Round %d, x after modExp: %llu\n", round, x[0]);

        if (x[0] == 1 || x[0] == n[0] - 1) continue; // Possibly prime

        int continueLoop = 0;
        for (int i = 1; i < r; i++) {
            modExp(x, x, n, x); // x = (x * x) % n
            if (x[0] == n[0] - 1) {
                continueLoop = 1;
                break;
            }
            // Debug: Print x during squaring
            printf("Inner loop %d, x during squaring: %llu\n", i, x[0]);
        }
        if (!continueLoop) {
            results[idx] = 0; // Composite
            return;
        }
    }
}

void initCurandWrapper(hiprandState *d_state, int numCandidates) {
    initCurand<<<(numCandidates + 255) / 256, 256>>>(d_state, time(0));
}

// Run Miller-Rabin primality test on a range of large numbers (in segments)
void millerRabinTestRange(uint64_t *numbers, int *results, int numCandidates) {
    hiprandState *d_state;
    hipMalloc(&d_state, numCandidates * sizeof(hiprandState));
    initCurandWrapper(d_state, numCandidates);

    uint64_t *d_numbers;
    int *d_results;
    hipMalloc(&d_numbers, numCandidates * NUM_SEGMENTS * sizeof(uint64_t));
    hipMalloc(&d_results, numCandidates * sizeof(int));

    hipMemcpy(d_numbers, numbers, numCandidates * NUM_SEGMENTS * sizeof(uint64_t), hipMemcpyHostToDevice);

    millerRabinTestKernel<<<(numCandidates + 255) / 256, 256>>>(d_numbers, d_results, d_state, numCandidates);

    hipMemcpy(results, d_results, numCandidates * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_state);
    hipFree(d_numbers);
    hipFree(d_results);
}
}
