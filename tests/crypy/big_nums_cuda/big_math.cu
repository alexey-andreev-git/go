// big_math.cu
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

typedef struct modExpParams {
    uint64_t base;
    uint64_t exp;
    uint64_t mod;
    uint64_t result;
} modExpParams;

// CUDA kernel for modular exponentiation (simple example)
__global__ void modExp(modExpParams *params, uint64_t numParams) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParams) {
        uint64_t res = 1;
        uint64_t b = params[idx].base;
        uint64_t exp = params[idx].exp;
        uint64_t mod = params[idx].mod;
        while (exp > 0) {
            if (exp % 2 == 1) {
                res = (res * b) % mod;
            }
            b = (b * b) % mod;
            exp /= 2;
        }
        params[idx].result = res;
    }
    // uint64_t res = 1;
    // uint64_t b = *base;
    // while (exp > 0) {
    //     if (exp % 2 == 1) {
    //         res = (res * b) % mod;
    //     }
    //     b = (b * b) % mod;
    //     exp /= 2;
    // }
    // *result = res;
}

// Wrapper function to call the kernel
extern "C" void cudaModExp(modExpParams *params, uint64_t numParams) {
    // uint64_t *d_base, *d_result;
    // cudaMalloc(&d_base, sizeof(uint64_t));
    // cudaMalloc(&d_result, sizeof(uint64_t));
    modExpParams *d_params;
    hipMalloc(&d_params, numParams * sizeof(modExpParams));
    
    // cudaMemcpy(d_base, base, sizeof(uint64_t), cudaMemcpyHostToDevice);
    hipMemcpy(d_params, params, numParams * sizeof(modExpParams), hipMemcpyHostToDevice);

    // modExp<<<1, 1>>>(d_base, exp, mod, d_result);  // Launch kernel on 1 block, 1 thread
    modExp<<<(numParams + 255) / 256, 256>>>(d_params, numParams);

    // cudaMemcpy(result, d_result, sizeof(uint64_t), cudaMemcpyDeviceToHost);
    hipMemcpy(params, d_params, numParams * sizeof(modExpParams), hipMemcpyDeviceToHost);

    // cudaFree(d_base);
    // cudaFree(d_result);
    hipFree(d_params);
}
